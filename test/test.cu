#include "hip/hip_runtime.h"
#include "hesai_lidar_sdk_gpu.cuh"
uint32_t last_frame_time;
uint32_t cur_frame_time;std::mutex mex_viewer;


//log info, display frame message
void lidarCallback(const LidarDecodedFrame<LidarPointXYZIRT>  &frame) {
  cur_frame_time = GetMicroTickCount();  
  if (cur_frame_time - last_frame_time > kMaxTimeInterval) {
    printf("Time between last frame and cur frame is: %d us\n", (cur_frame_time - last_frame_time));
  }
  last_frame_time = cur_frame_time;    
  printf("frame:%d points:%u packet:%d start time:%lf end time:%lf\n",frame.frame_index, frame.points_num, frame.packet_num, frame.points[0].timestamp, frame.points[frame.points_num - 1].timestamp) ;
}


int main(int argc, char *argv[])
{
  HesaiLidarSdkGpu<LidarPointXYZIRT> sample;
  DriverParam param;
  // assign param
  param.decoder_param.enable_parser_thread = false;
  param.input_param.source_type = DATA_FROM_LIDAR;
  param.input_param.pcap_path = "";
  param.input_param.correction_file_path = "";
  param.input_param.firetimes_path = "";

  param.input_param.device_ip_address = "192.168.123.20";
  param.input_param.ptc_port = 9347;
  param.input_param.udp_port = 2368;
  param.input_param.host_ip_address = "192.168.123.24";
  param.input_param.multicast_ip_address = "";



  //init lidar with param
  sample.Init(param);
  float socket_buffer = 262144000;
  sample.lidar_ptr_->source_->SetSocketBufferSize(socket_buffer);

  //assign callback fuction
  sample.RegRecvCallback(lidarCallback);

  //star process thread
  sample.Start();
  while (1)
  {
  
    std::this_thread::sleep_for(std::chrono::milliseconds(40));

  }
}